/***
 * Author:Yun-Chen Lo
 * File:HW4_cuda.cu
 */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <assert.h>

#define INF 1000000000


int *Hostmap; 	// host memory for input adjacent file
int *devMap;	// device memory
int N; 			// number of vertex

void readInput(const char* infile){
	FILE * pFile;
	int in, counter=0;
	int i, j;
	pFile = fopen ( infile , "r" );
	fscanf (pFile, "%d", &in);
	N = in;
	hipHostMalloc(&Hostmap, (N*N)*sizeof(int), hipHostMallocDefault);
	for(i=0; i<N; i++){
  		for(j=0; j<N; j++){
  			if(i!=j)
  				Hostmap[N*i + j] = INF;
  			else
  				Hostmap[N*i + j] = 0;
  		}
  	}
  	while (!feof (pFile))
    {  
		fscanf (pFile, "%d", &in); 
		counter ++;
		if(counter > 1){
			if((counter-2) % 3 == 0){
				i=in;
			}
			else if ((counter-2) % 3 == 1 ){
				j=in;
			}
			else if((counter-2) % 3 == 2){
				Hostmap[N*i + j] = in;
			}
      	}
    }
}

int ceil(int a, int b){
	return (a + b -1)/b;
}

__global__ void floyd_phaseI(int k, int *devMap, int B, int d_N){
	__shared__ int shared_mem[32][32];
	int i = k * B + threadIdx.y;
	int j = k * B + threadIdx.x;
	if(i < d_N && j < d_N){
		int g_mem_index = i * d_N + j;
		shared_mem[i][j] = devMap[g_mem_index];
		__syncthreads();

		for(int l = 0; l < B; l++){
			if (shared_mem[i][l] + shared_mem[l][j] < shared_mem[i][j]){
				shared_mem[i][j] = shared_mem[i][l] + shared_mem[l][j];
			}
		}
		devMap[g_mem_index] = shared_mem[i][j];
	}
}
// add memory boundary
__global__ void floyd_phaseII(int k, int *devMap, int B, int d_N){
	if(blockIdx.x != k){
		__shared__ int shared_mem[32][32], shared_buffer[32][32];
		int i = k * B + threadIdx.y;
		int j = k * B + threadIdx.x;
		int g_mem_index = i * d_N + j;
		shared_mem[i][j] = devMap[g_mem_index];

		if(blockIdx.y == 0){ 	// row
			i = k * B + threadIdx.y;
			j = B * blockIdx.x + threadIdx.x;
		}
		else { 					// col
			i = B * blockIdx.x + threadIdx.y;
			j = k * B + threadIdx.x;
		}
		g_mem_index = i * d_N + j;
		shared_buffer[i][j] = devMap[g_mem_index];
		__syncthreads();

		if(blockIdx.y == 0){
			for(int l = 0; l < B; l++){
				if(shared_mem[i][l] + shared_buffer[l][j] < shared_buffer[i][j]){
					shared_buffer[i][j] = shared_mem[i][l] + shared_buffer[l][j];
				}
			}
		}
		else{
			for(int l = 0; l < B; l++){
				if(shared_buffer[i][l] + shared_mem[l][j] < shared_buffer[i][j]){
					shared_buffer[i][j] = shared_mem[i][l] + shared_buffer[l][j];
				}
			}
		}
		devMap[g_mem_index] = shared_buffer[i][j];
	}
}

__global__ void floyd_phaseIII(int k, int *devMap, int B, int d_N){
	if(blockIdx.x!= k && blockIdx.y!= k){
		__shared__ int d_c[32][32], d_r[32][32];
		int base = k * B;
		int d_i = blockDim.y * blockIdx.y + threadIdx.y;
		int d_j = blockDim.x * blockIdx.x + threadIdx.x;
		int i = threadIdx.y;
		int j = threadIdx.x;
		int col_base = (base + i) * d_N + d_j;
		int row_base = d_i * d_N + base + j;
		base = d_i * d_N + d_j;
		d_r[i][j] = devMap[col_base];
		d_c[i][j] = devMap[row_base];
		int oldD = devMap[base];
		__syncthreads();

		int newD;
		for (int t = 0; t < B; t++) {
			newD = d_c[i][t] + d_r[t][j];
			if (newD < oldD)
				oldD = newD;
		}
		devMap[base] = oldD;
	}
}

void Block_floydWarshall(int* devMap, int B){
	int k;
	int round = ceil(N, B);
	int BLKSZ;
	if(round == 1){
    	BLKSZ = N;
    }
    else{
    	BLKSZ = round;
    }
    dim3 blockSize1(BLKSZ, BLKSZ);

    dim3 gridSize2(N / BLKSZ, 2);
    dim3 blockSize2(BLKSZ, BLKSZ);

	dim3 gridSize3(N / BLKSZ, N / BLKSZ);
	dim3 blockSize3(BLKSZ, BLKSZ);
    
    int d_N = N;

    printf("N/BLKSZ = %d",N/BLKSZ);
    printf("round = %d\n", round);
    for(k = 0; k<round; k++){
    	floyd_phaseI<<<1, blockSize1>>>(k, devMap, BLKSZ, d_N);
    	//floyd_phaseII<<<gridSize2, blockSize2>>>(k, devMap, BLKSZ, d_N);
    	//floyd_phaseIII<<<gridSize3, blockSize3>>>(k, devMap, BLKSZ, d_N);
    }
}

void saveSolution(const char* outfile){
	FILE *out;
	out=fopen(outfile, "wb");
	fwrite(Hostmap,sizeof(int),N*N,out);
    fclose(out);
}

int main(int argc, char** argv) {
	const char* infile = argv[1];
	const char* outfile = argv[2];
	int B = atoi(argv[3]);	//block size
	readInput(infile);
	hipMalloc(&devMap, N * N * sizeof(int));
	hipMemcpy(devMap, Hostmap, sizeof(int) * N * N, hipMemcpyHostToDevice);
	Block_floydWarshall(devMap, B);
	hipMemcpy(Hostmap, devMap, sizeof(int) * N * N, hipMemcpyDeviceToHost);
	printf("%d %d %d \n", Hostmap[0], Hostmap[1], Hostmap[2]);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) 
		    printf("Error: %s\n", hipGetErrorString(err));
	saveSolution(outfile);
	return 0;
}