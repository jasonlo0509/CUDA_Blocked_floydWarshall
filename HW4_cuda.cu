/***
 * Author:Yun-Chen Lo
 * File:HW4_cuda.cu
 */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <assert.h>


#define INF 1000000000


int *Hostmap; 	// host memory for input adjacent file
int *devMap;	// device memory
int N; 			// number of vertex


int ceil(int a, int b){
	return (a + b -1)/b;
}

int readInput(const char* infile, int B){
	FILE * pFile;
	int in;
	int i, j, width;
	pFile = fopen ( infile , "r" );
	int m;
	fscanf(pFile, "%d %d", &N, &m);
	if(ceil(N, B) == N/B || ceil(N, B) == 1){
		width = N;
		hipHostMalloc(&Hostmap, (width*width)*sizeof(int), hipHostMallocDefault);
	}
	else{
		width = ceil(N, B) * B;
		hipHostMalloc(&Hostmap, (width*width)*sizeof(int), hipHostMallocDefault);
	}
	printf("width = %d\n ", width);
	for(i=0; i<width; i++){
  		for(j=0; j<width; j++){
  			if(i!=j)
  				Hostmap[width*i + j] = INF;
  			else if(i == j && i < N)
  				Hostmap[width*i + j] = 0;
  			else
  				Hostmap[width*i + j] = INF;
  		}
  	}
  	while (--m >= 0)
    {  
		fscanf(pFile, "%d %d %d", &i, &j, &in);
		Hostmap[width*i + j] = in;
    }
    return width;
}


__global__ void floyd_phaseI(int k, int *devMap, int B, int d_N){
	extern __shared__ int shared_mem[];
	int i = threadIdx.y;
	int j = threadIdx.x;
	int base = k * B;
	int d_i = base + i;
	int d_j = base + j;
	int g_mem_index = d_i * d_N + d_j;
	shared_mem[i*B + j] = devMap[g_mem_index];
	__syncthreads();
	#pragma unroll 16
	for(int l = 0; l < B; l++){
		if (shared_mem[i*B + l] + shared_mem[l*B + j] < shared_mem[i*B + j]){
			shared_mem[i*B + j] = shared_mem[i*B + l] + shared_mem[l*B + j];
		}
		__syncthreads();
	}
	devMap[g_mem_index] = shared_mem[i*B + j];
}

__global__ void floyd_phaseII(int k, int *devMap, int B, int d_N){
	extern __shared__ int S[];
	if(blockIdx.x != k){
		int *shared_mem = &S[0];
		int *shared_buffer = &S[B*B];
		int i = threadIdx.y;
		int j = threadIdx.x;
		int base = k*B;
		int d_i, d_j;
		int g_mem_index = (i+base) * d_N + (j+base);
		if(blockIdx.y == 0){ 	// row
			d_i = base + threadIdx.y;
			d_j = blockDim.x * blockIdx.x + threadIdx.x;
			shared_mem[j*B + i] = devMap[g_mem_index];
			g_mem_index = d_i * d_N + d_j;
			shared_buffer[i*B + j] = devMap[g_mem_index];
		}
		else { 					// col
			d_i = blockDim.x * blockIdx.x + threadIdx.y;
			d_j = base + threadIdx.x;
			shared_mem[i*B + j] = devMap[g_mem_index];
			g_mem_index = d_i * d_N + d_j;
			shared_buffer[j*B + i] = devMap[g_mem_index];
		}
		__syncthreads();

		if(blockIdx.y == 0){
			#pragma unroll 16
			for(int l = 0; l < B; l++){
				if(shared_mem[l*B + i] + shared_buffer[l*B + j] < shared_buffer[i*B +j]){
					shared_buffer[i*B + j] = shared_mem[l*B +i] + shared_buffer[l*B +j];
				}
				__syncthreads();
			}
			devMap[g_mem_index] = shared_buffer[i*B +j];
		}
		else{
			#pragma unroll 16
			for(int l = 0; l < B; l++){
				if(shared_buffer[l*B +i] + shared_mem[l*B +j] < shared_buffer[j*B +i]){
					shared_buffer[j*B +i] = shared_buffer[l*B +i] + shared_mem[l*B +j];
				}
				__syncthreads();
			}
			devMap[g_mem_index] = shared_buffer[j*B +i];
		}
	}
}

__global__ void floyd_phaseIII(int k, int *devMap, int B, int d_N){
	extern __shared__ int S[];
	if(blockIdx.x!= k && blockIdx.y!= k){
		//__shared__ int d_c[16][16], d_r[16][16];
		int *d_c = &S[0];
		int *d_r = &S[B*B];
		int base = k * B;
		int d_i = blockDim.y * blockIdx.y + threadIdx.y;
		int d_j = blockDim.x * blockIdx.x + threadIdx.x;
		int i = threadIdx.y;
		int j = threadIdx.x;
		int col_base = (base + i) * d_N + d_j;
		int row_base = d_i * d_N + (base + j);
		base = d_i * d_N + d_j;
		d_r[i*B + j] = devMap[col_base];
		d_c[j*B + i] = devMap[row_base];
		int oldD = devMap[base];
		__syncthreads();

		int newD;
		#pragma unroll 16
		for (int t = 0; t < B; t++) {
			newD = d_c[t*B + i] + d_r[t*B + j];
			if (newD < oldD)
				oldD = newD;
			__syncthreads();
		}
		devMap[base] = oldD;
	}
}

void Block_floydWarshall(int* devMap, int B, int width){
	int k;
	int round = ceil(N, B);
	int BLKSIZE;
	if(round == 1){
    	BLKSIZE = N;
    }
    else{
    	BLKSIZE = B;
    }
    dim3 blockSize1(BLKSIZE, BLKSIZE);

    dim3 gridSize2(round, 2);
    dim3 blockSize2(BLKSIZE, BLKSIZE);

	dim3 gridSize3(round, round);
	dim3 blockSize3(BLKSIZE, BLKSIZE);

    printf("BLKSIZE = %d",BLKSIZE);
    printf("round = %d\n", round);
    for(k = 0; k<round; k++){
    	floyd_phaseI<<<1, blockSize1, B*B*sizeof(int)>>>(k, devMap, BLKSIZE, width);
    	floyd_phaseII<<<gridSize2, blockSize2, 2*B*B*sizeof(int)>>>(k, devMap, BLKSIZE, width);
    	floyd_phaseIII<<<gridSize3, blockSize3, 2*B*B*sizeof(int)>>>(k, devMap, BLKSIZE, width);
    }
}

void saveSolution(int* FinalMap, const char* outfile){
	FILE *out;
	out=fopen(outfile, "wb");
	fwrite(FinalMap,sizeof(int),N*N,out);
    fclose(out);
}

int main(int argc, char** argv) {
	const char* infile = argv[1];
	const char* outfile = argv[2];
	int B = atoi(argv[3]);	//block size
	int width;
	width = readInput(infile, B);

	hipMalloc(&devMap, width * width * sizeof(int));
	hipMemcpy(devMap, Hostmap, sizeof(int) * width * width, hipMemcpyHostToDevice);

	Block_floydWarshall(devMap, B, width);
	
	hipMemcpy(Hostmap, devMap, sizeof(int) * width * width, hipMemcpyDeviceToHost);
	hipFree(devMap);

	printf("Hostmap\n");
	printf("%d %d\n", Hostmap[1], Hostmap[width*width-1]);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) 
		    printf("Error: %s\n", hipGetErrorString(err));
	int *FinalMap = (int*)malloc((N*N)*sizeof(int));
	for(int i = 0; i < width; i++){
		for(int j = 0; j < width; j++){
			if(i < N && j < N)
				FinalMap[N * i + j] = Hostmap[width*i + j];
		}
	}
	saveSolution(FinalMap, outfile);
	return 0;
}