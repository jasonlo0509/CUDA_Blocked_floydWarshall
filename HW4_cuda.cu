/***
 * Author:Yun-Chen Lo
 * File:cuda_debug.cu
 */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <assert.h>

#define INF 1000000000


int *Hostmap; 	// host memory for input adjacent file
int *devMap;	// device memory
int N; 			// number of vertex


int ceil(int a, int b){
	return (a + b -1)/b;
}

int readInput(const char* infile, int B){
	FILE * pFile;
	int in;
	int i, j, width;
	pFile = fopen ( infile , "r" );
	int m;
	fscanf(pFile, "%d %d", &N, &m);
	if(ceil(N, B) == N/B || ceil(N, B) == 1){
		width = N;
		hipHostMalloc(&Hostmap, (width*width)*sizeof(int), hipHostMallocDefault);
	}
	else{
		width = ceil(N, B) * B;
		hipHostMalloc(&Hostmap, (width*width)*sizeof(int), hipHostMallocDefault);
	}
	printf("width = %d\n ", width);
	for(i=0; i<width; i++){
  		for(j=0; j<width; j++){
  			if(i!=j)
  				Hostmap[width*i + j] = INF;
  			else if(i == j && i < N)
  				Hostmap[width*i + j] = 0;
  			else
  				Hostmap[width*i + j] = INF;
  		}
  	}
  	while (--m >= 0)
    {  
		fscanf(pFile, "%d %d %d", &i, &j, &in);
		Hostmap[width*i + j] = in;
    }
    return width;
}


__global__ void floyd_phaseI(int k, int *devMap, int B, int d_N){
	__shared__ int shared_mem[16][16];
	int i = threadIdx.y;
	int j = threadIdx.x;
	int base = k * B;
	int d_i = base + i;
	int d_j = base + j;
	int g_mem_index = d_i * d_N + d_j;
	shared_mem[i][j] = devMap[g_mem_index];
	__syncthreads();
	#pragma unroll 16
	for(int l = 0; l < B; l++){
		if (shared_mem[i][l] + shared_mem[l][j] < shared_mem[i][j]){
			shared_mem[i][j] = shared_mem[i][l] + shared_mem[l][j];
		}
		__syncthreads();
	}
	devMap[g_mem_index] = shared_mem[i][j];
}

__global__ void floyd_phaseII(int k, int *devMap, int B, int d_N){
	if(blockIdx.x != k){
		__shared__ int shared_mem[16][16], shared_buffer[16][16];
		int i = threadIdx.y;
		int j = threadIdx.x;
		int base = k*B;
		int d_i, d_j;
		int g_mem_index = (i+base) * d_N + (j+base);
		if(blockIdx.y == 0){ 	// row
			d_i = base + threadIdx.y;
			d_j = blockDim.x * blockIdx.x + threadIdx.x;
			shared_mem[j][i] = devMap[g_mem_index];
			g_mem_index = d_i * d_N + d_j;
			shared_buffer[i][j] = devMap[g_mem_index];
		}
		else { 					// col
			d_i = blockDim.x * blockIdx.x + threadIdx.y;
			d_j = base + threadIdx.x;
			shared_mem[i][j] = devMap[g_mem_index];
			g_mem_index = d_i * d_N + d_j;
			shared_buffer[j][i] = devMap[g_mem_index];
		}
		__syncthreads();

		if(blockIdx.y == 0){
			#pragma unroll 16
			for(int l = 0; l < B; l++){
				if(shared_mem[l][i] + shared_buffer[l][j] < shared_buffer[i][j]){
					shared_buffer[i][j] = shared_mem[l][i] + shared_buffer[l][j];
				}
				__syncthreads();
			}
			devMap[g_mem_index] = shared_buffer[i][j];
		}
		else{
			#pragma unroll 16
			for(int l = 0; l < B; l++){
				if(shared_buffer[l][i] + shared_mem[l][j] < shared_buffer[j][i]){
					shared_buffer[j][i] = shared_buffer[l][i] + shared_mem[l][j];
				}
				__syncthreads();
			}
			devMap[g_mem_index] = shared_buffer[j][i];
		}
	}
}

__global__ void floyd_phaseIII(int k, int *devMap, int B, int d_N){
	if(blockIdx.x!= k && blockIdx.y!= k){
		__shared__ int d_c[16][16], d_r[16][16];
		int base = k * B;
		int d_i = blockDim.y * blockIdx.y + threadIdx.y;
		int d_j = blockDim.x * blockIdx.x + threadIdx.x;
		int i = threadIdx.y;
		int j = threadIdx.x;
		int col_base = (base + i) * d_N + d_j;
		int row_base = d_i * d_N + (base + j);
		base = d_i * d_N + d_j;
		d_r[i][j] = devMap[col_base];
		d_c[j][i] = devMap[row_base];
		int oldD = devMap[base];
		__syncthreads();

		int newD;
		#pragma unroll 16
		for (int t = 0; t < B; t++) {
			newD = d_c[t][i] + d_r[t][j];
			if (newD < oldD)
				oldD = newD;
			__syncthreads();
		}
		devMap[base] = oldD;
	}
}

void Block_floydWarshall(int* devMap, int B, int width){
	int k;
	int round = ceil(N, B);
	int BLKSIZE;
	if(round == 1){
    	BLKSIZE = N;
    }
    else{
    	BLKSIZE = B;
    }
    dim3 blockSize1(BLKSIZE, BLKSIZE);

    dim3 gridSize2(round, 2);
    dim3 blockSize2(BLKSIZE, BLKSIZE);

	dim3 gridSize3(round, round);
	dim3 blockSize3(BLKSIZE, BLKSIZE);

    printf("BLKSIZE = %d",BLKSIZE);
    printf("round = %d\n", round);
    for(k = 0; k<round; k++){
    	floyd_phaseI<<<1, blockSize1>>>(k, devMap, BLKSIZE, width);
    	floyd_phaseII<<<gridSize2, blockSize2>>>(k, devMap, BLKSIZE, width);
    	floyd_phaseIII<<<gridSize3, blockSize3>>>(k, devMap, BLKSIZE, width);
    }
}

void saveSolution(int* FinalMap, const char* outfile){
	FILE *out;
	out=fopen(outfile, "wb");
	fwrite(FinalMap,sizeof(int),N*N,out);
    fclose(out);
}

int main(int argc, char** argv) {
	const char* infile = argv[1];
	const char* outfile = argv[2];
	int B = atoi(argv[3]);	//block size
	int width;
	width = readInput(infile, B);
	hipMalloc(&devMap, width * width * sizeof(int));
	hipMemcpy(devMap, Hostmap, sizeof(int) * width * width, hipMemcpyHostToDevice);
	Block_floydWarshall(devMap, B, width);
	hipMemcpy(Hostmap, devMap, sizeof(int) * width * width, hipMemcpyDeviceToHost);
	hipFree(devMap);
	/*for(int i = 0; i < width*width; i++)
		printf("%d ", Hostmap[i]);
	printf("\n");
	*/
	printf("Hostmap\n");
	printf("%d %d\n", Hostmap[1], Hostmap[width*width-1]);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) 
		    printf("Error: %s\n", hipGetErrorString(err));
	int *FinalMap;
	hipHostMalloc(&FinalMap, (N*N)*sizeof(int), hipHostMallocDefault);
	for(int i = 0; i < width; i++){
		for(int j = 0; j < width; j++){
			if(i < N && j < N)
				FinalMap[N * i + j] = Hostmap[width*i + j];
		}
	}
	saveSolution(FinalMap, outfile);
	return 0;
}