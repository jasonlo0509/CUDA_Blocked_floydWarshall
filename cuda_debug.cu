/***
 * Author:Yun-Chen Lo
 * File:cuda_debug.cu
 */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <assert.h>

#define INF 1000000000


int *Hostmap; 	// host memory for input adjacent file
int *devMap;	// device memory
int N; 			// number of vertex


int ceil(int a, int b){
	return (a + b -1)/b;
}

int readInput(const char* infile, int B){
	FILE * pFile;
	int in, counter=0;
	int i, j, width;
	pFile = fopen ( infile , "r" );
	fscanf (pFile, "%d", &in);
	N = in;
	if(ceil(N, B) == N/B ){
		width = N;
		hipHostMalloc(&Hostmap, (width*width)*sizeof(int), hipHostMallocDefault);
	}
	else{
		width = ceil(N, B) * B;
		hipHostMalloc(&Hostmap, (width*width)*sizeof(int), hipHostMallocDefault);
	}
	printf("width = %d\n ", width);
	for(i=0; i<width; i++){
  		for(j=0; j<width; j++){
  			if(i!=j)
  				Hostmap[width*i + j] = INF;
  			else if(i == j && i < N)
  				Hostmap[width*i + j] = 0;
  			else
  				Hostmap[width*i + j] = INF;
  		}
  	}
  	while (!feof (pFile))
    {  
		fscanf (pFile, "%d", &in); 
		counter ++;
		if(counter > 1){
			if((counter-2) % 3 == 0){
				i=in;
			}
			else if ((counter-2) % 3 == 1 ){
				j=in;
			}
			else if((counter-2) % 3 == 2){
				Hostmap[width*i + j] = in;
			}
      	}
    }
    return width;
}


__global__ void floyd_phaseI(int k, int *devMap, int B, int d_N){
	__shared__ int shared_mem[32][32];
	int i = threadIdx.y;
	int j = threadIdx.x;
	int d_i = k * B + i;
	int d_j = k * B + j;
	int g_mem_index = d_i * d_N + d_j;
	shared_mem[i][j] = devMap[g_mem_index];
	__syncthreads();

	for(int l = 0; l < B; l++){
		if (shared_mem[i][l] + shared_mem[l][j] < shared_mem[i][j]){
			shared_mem[i][j] = shared_mem[i][l] + shared_mem[l][j];
		}
		__syncthreads();
	}
	devMap[g_mem_index] = shared_mem[i][j];
}

__global__ void floyd_phaseII(int k, int *devMap, int B, int d_N){
	if(blockIdx.x != k){
		__shared__ int shared_mem[32][32], shared_buffer[32][32];
		int i = threadIdx.y;
		int j = threadIdx.x;
		int d_i, d_j;
		if(blockIdx.y == 0){ 	// row
			d_i = k * B + threadIdx.y;
			d_j = blockDim.x * blockIdx.x + threadIdx.x;//problem
		}
		else { 					// col
			d_i = blockDim.x * blockIdx.x + threadIdx.y;
			d_j = k * B + threadIdx.x;
		}
		
		int g_mem_index = (i+B*k) * d_N + (j+B*k);
		shared_mem[i][j] = devMap[g_mem_index];
		g_mem_index = d_i * d_N + d_j;
		shared_buffer[i][j] = devMap[g_mem_index];
		__syncthreads();

		if(blockIdx.y == 0){
			for(int l = 0; l < B; l++){
				if(shared_mem[i][l] + shared_buffer[l][j] < shared_buffer[i][j]){
					shared_buffer[i][j] = shared_mem[i][l] + shared_buffer[l][j];
				}
				__syncthreads();
			}
		}
		else{
			for(int l = 0; l < B; l++){
				if(shared_buffer[i][l] + shared_mem[l][j] < shared_buffer[i][j]){
					shared_buffer[i][j] = shared_buffer[i][l] + shared_mem[l][j];
				}
				__syncthreads();
			}
		}
		devMap[g_mem_index] = shared_buffer[i][j];

	}
}

__global__ void floyd_phaseIII(int k, int *devMap, int B, int d_N){
	if(blockIdx.x!= k && blockIdx.y!= k){
		__shared__ int d_c[32][32], d_r[32][32];
		int base = k * B;
		int d_i = blockDim.y * blockIdx.y + threadIdx.y;
		int d_j = blockDim.x * blockIdx.x + threadIdx.x;
		int i = threadIdx.y;
		int j = threadIdx.x;
		int col_base = (base + i) * d_N + d_j;
		int row_base = d_i * d_N + (base + j);
		base = d_i * d_N + d_j;
		d_r[i][j] = devMap[col_base];
		d_c[i][j] = devMap[row_base];
		int oldD = devMap[base];
		__syncthreads();

		int newD;
		for (int t = 0; t < B; t++) {
			newD = d_c[i][t] + d_r[t][j];
			if (newD < oldD)
				oldD = newD;
			__syncthreads();
		}
		devMap[base] = oldD;
	}
}

void Block_floydWarshall(int* devMap, int B, int width){
	int k;
	int round = ceil(N, B);
	int BLKSIZE;
	if(round == 1){
    	BLKSIZE = N;
    }
    else{
    	BLKSIZE = B;
    }
    dim3 blockSize1(BLKSIZE, BLKSIZE);

    dim3 gridSize2(round, 2);
    dim3 blockSize2(BLKSIZE, BLKSIZE);

	dim3 gridSize3(round, round);
	dim3 blockSize3(BLKSIZE, BLKSIZE);
    
    //int d_N = N;

    printf("BLKSIZE = %d",BLKSIZE);
    printf("round = %d\n", round);
    for(k = 0; k<round; k++){
    	floyd_phaseI<<<1, blockSize1>>>(k, devMap, BLKSIZE, width);
    	floyd_phaseII<<<gridSize2, blockSize2>>>(k, devMap, BLKSIZE, width);
    	floyd_phaseIII<<<gridSize3, blockSize3>>>(k, devMap, BLKSIZE, width);
    }
}

void saveSolution(int* FinalMap, const char* outfile){
	FILE *out;
	out=fopen(outfile, "wb");
	fwrite(FinalMap,sizeof(int),N*N,out);
    fclose(out);
}

int main(int argc, char** argv) {
	const char* infile = argv[1];
	const char* outfile = argv[2];
	int B = atoi(argv[3]);	//block size
	int width;
	width = readInput(infile, B);
	hipMalloc(&devMap, width * width * sizeof(int));
	hipMemcpy(devMap, Hostmap, sizeof(int) * width * width, hipMemcpyHostToDevice);
	Block_floydWarshall(devMap, B, width);
	hipMemcpy(Hostmap, devMap, sizeof(int) * width * width, hipMemcpyDeviceToHost);
	hipFree(devMap);
	for(int i = 0; i < width*width; i++)
		printf("%d ", Hostmap[i]);
	printf("\n");
	
	printf("Hostmap\n");
	printf("%d %d\n", Hostmap[1], Hostmap[width*width-1]);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) 
		    printf("Error: %s\n", hipGetErrorString(err));
	int *FinalMap;
	hipHostMalloc(&FinalMap, (N*N)*sizeof(int), hipHostMallocDefault);
	for(int i = 0; i < width; i++){
		for(int j = 0; j < width; j++){
			if(i < N && j < N)
				FinalMap[N * i + j] = Hostmap[width*i + j];
		}
	}
	saveSolution(FinalMap, outfile);
	return 0;
}